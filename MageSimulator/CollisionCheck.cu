#include "hip/hip_runtime.h"
#include "CollisionCheck.cuh"



hipError_t checkCollisions(int cnt, Mesh* mesh, Ray* ray, CollisionSet* col)
{
  if (cnt > 0)
  {
    hipError_t cudaStatus;
    Mesh* dev_mesh;
    Ray* dev_ray;
    CollisionSet* dev_col;

    // Choose which GPU to run on, change this on a multi-GPU system.
    //cudaStatus = hipSetDevice(0);
    //if (cudaStatus != hipSuccess) {
    //  fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    //  goto Error4;
    //}

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_mesh, cnt * sizeof(Mesh));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed_1!");
      goto Error3;
    }
    cudaStatus = hipMalloc((void**)&dev_ray, cnt * sizeof(Ray));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed_2!");
      goto Error2;
    }
    cudaStatus = hipMalloc((void**)&dev_col, cnt * sizeof(CollisionSet));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed_3!");
      goto Error1;
    }

    // create stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Copy input vectors from host memory to GPU buffers.
    //cudaStatus = hipMemcpy(dev_mesh, mesh, cnt * sizeof(Mesh), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpyAsync(dev_mesh, mesh, cnt * sizeof(Mesh), hipMemcpyHostToDevice, stream);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed_1!"); 
      goto Error1;
    }
    //cudaStatus = hipMemcpy(dev_ray, ray, cnt * sizeof(Ray), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpyAsync(dev_ray, ray, cnt * sizeof(Ray), hipMemcpyHostToDevice, stream);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed_2!");
      goto Error1;
    }

    // Launch a kernel on the GPU with one thread for each element.
    checkCollision << <(cnt + 1024 -1) / 1024, 1024 >> > (cnt, dev_mesh, dev_ray, dev_col);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error1;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //  fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //  
    //  hipError_t err = hipGetLastError();
    //  if (err != hipSuccess) {
    //    printf("CUDA error: %s\n", hipGetErrorString(err));
    //  }
    //  
    //  goto Error1;
    //}

    // Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(col, dev_col, cnt * sizeof(CollisionSet), hipMemcpyDeviceToHost);
    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    //hipEventRecord(start, 0);
    cudaStatus = hipMemcpyAsync(col, dev_col, cnt * sizeof(CollisionSet), hipMemcpyDeviceToHost, stream);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed!");
      goto Error1;
    }
    //hipEventRecord(stop, 0);
    // delete stream
    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipStreamSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
      
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
      }
      
      goto Error1;
    }

    hipStreamDestroy(stream);
    //hipEventSynchronize(stop);
    //float elapsedTime;
    //hipEventElapsedTime(&elapsedTime, start, stop);
    //float bandwidth = (cnt * sizeof(CollisionSet) / elapsedTime) * 1e3;
    //printf("%lf\n", bandwidth);

  Error1:
    hipFree(dev_col);
  Error2:
    hipFree(dev_ray);
  Error3:
    hipFree(dev_mesh);
  Error4:
    return cudaStatus;
  }
  else
  {
    return hipSuccess;
  }
}

__device__ float det(float a[3], float b[3], float c[3])
{
  return (a[0] * b[1] * c[2] + a[1] * b[2] * c[0] + a[2] * b[0] * c[1]
    - a[0] * b[2] * c[1] - a[1] * b[0] * c[2] - a[2] * b[1] * c[0]);
}

//https://shikousakugo.wordpress.com/2012/07/01/ray-intersection-3/
__global__ void checkCollision(int cnt, Mesh* _mesh, Ray* _ray, CollisionSet* _col)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < cnt)
  {
    Mesh targetMesh = _mesh[idx];
    Ray targetRay = _ray[idx];
    _col[idx].meshId = targetMesh.id;
    _col[idx].rayId = targetRay.id;
    _col[idx].colFlag = false;

    // calc
    float ray[3];
    ray[0] = targetRay.tar[0] - targetRay.org[0];
    ray[1] = targetRay.tar[1] - targetRay.org[1];
    ray[2] = targetRay.tar[2] - targetRay.org[2];
    float rayLen = sqrt(ray[0] * ray[0] + ray[1] * ray[1] + ray[2] * ray[2]);
    float edge1[3];
    edge1[0] = targetMesh.pnt1[0] - targetMesh.pnt0[0];
    edge1[1] = targetMesh.pnt1[1] - targetMesh.pnt0[1];
    edge1[2] = targetMesh.pnt1[2] - targetMesh.pnt0[2];
    float edge1Len = sqrt(edge1[0] * edge1[0] + edge1[1] * edge1[1] + edge1[2] * edge1[2]);
    float edge2[3];
    edge2[0] = targetMesh.pnt2[0] - targetMesh.pnt0[0];
    edge2[1] = targetMesh.pnt2[1] - targetMesh.pnt0[1];
    edge2[2] = targetMesh.pnt2[2] - targetMesh.pnt0[2];
    float edge2Len = sqrt(edge2[0] * edge2[0] + edge2[1] * edge2[1] + edge2[2] * edge2[2]);
    float p[3];
    p[0] = ray[1] * edge2[2] - ray[2] * edge2[1];
    p[1] = ray[2] * edge2[0] - ray[0] * edge2[2];
    p[2] = ray[0] * edge2[1] - ray[1] * edge2[0];

    float det = p[0] * edge1[0] + p[1] * edge1[1] + p[2] * edge1[2];
    if (det > EPSION)
    {
      float vt[3];
      vt[0] = targetRay.org[0] - targetMesh.pnt0[0];
      vt[1] = targetRay.org[1] - targetMesh.pnt0[1];
      vt[2] = targetRay.org[2] - targetMesh.pnt0[2];
      float u = p[0] * vt[0] + p[1] * vt[1] + p[2] * vt[2];
      if (u >= 0.0 && u <= 1.0 * det)
      {
        float q[3];
        q[0] = vt[1] * edge1[2] - vt[2] * edge1[1];
        q[1] = vt[2] * edge1[0] - vt[0] * edge1[2];
        q[2] = vt[0] * edge1[1] - vt[1] * edge1[0];
        float v = q[0] * ray[0] + q[1] * ray[1] + q[2] * ray[2];
        if (v >= 0.0 && (u + v) <= 1.0 * det)
        {
          float t = (q[0] * edge2[0] + q[1] * edge2[1] + q[2] * edge2[2]) / det;
          float pnt[3];
          u /= det;
          v /= det;
          pnt[0] = edge1[0] * u + edge2[0] * v + targetMesh.pnt0[0];
          pnt[1] = edge1[1] * u + edge2[1] * v + targetMesh.pnt0[1];
          pnt[2] = edge1[2] * u + edge2[2] * v + targetMesh.pnt0[2];
          float dist1 = sqrt((pnt[0] - targetRay.org[0]) * (pnt[0] - targetRay.org[0]) +
            (pnt[1] - targetRay.org[1]) * (pnt[1] - targetRay.org[1]) + 
            (pnt[2] - targetRay.org[2]) * (pnt[2] - targetRay.org[2]));
          if (t >= 0.0 && t <= 1.0)
          {
            // distance
            float cross[3];
            cross[0] = edge1[1] * edge2[2] - edge1[2] * edge2[1];
            cross[1] = edge1[2] * edge2[0] - edge1[0] * edge2[2];
            cross[2] = edge1[0] * edge2[1] - edge1[1] * edge2[0];
            float d = -cross[0] * targetMesh.pnt0[0] - cross[1] * targetMesh.pnt0[1] - cross[2] * targetMesh.pnt0[2];
            float dist2 = (cross[0] * pnt[0] + cross[1] * pnt[1] + cross[2] * pnt[2] + d) /
              sqrt(cross[0] * cross[0] + cross[1] * cross[1] + cross[2] * cross[2]);
            // output
            _col[idx].colFlag = true;
            _col[idx].pnt[0] = pnt[0];
            _col[idx].pnt[1] = pnt[1];
            _col[idx].pnt[2] = pnt[2];
          }
          else
          {
            // calc
            ray[0] = targetRay.org[0] - targetRay.tar[0];
            ray[1] = targetRay.org[1] - targetRay.tar[1];
            ray[2] = targetRay.org[2] - targetRay.tar[2];
            rayLen = sqrt(ray[0] * ray[0] + ray[1] * ray[1] + ray[2] * ray[2]);
            edge1[0] = targetMesh.pnt1[0] - targetMesh.pnt0[0];
            edge1[1] = targetMesh.pnt1[1] - targetMesh.pnt0[1];
            edge1[2] = targetMesh.pnt1[2] - targetMesh.pnt0[2];
            edge1Len = sqrt(edge1[0] * edge1[0] + edge1[1] * edge1[1] + edge1[2] * edge1[2]);
            edge2[0] = targetMesh.pnt2[0] - targetMesh.pnt0[0];
            edge2[1] = targetMesh.pnt2[1] - targetMesh.pnt0[1];
            edge2[2] = targetMesh.pnt2[2] - targetMesh.pnt0[2];
            edge2Len = sqrt(edge2[0] * edge2[0] + edge2[1] * edge2[1] + edge2[2] * edge2[2]);
            p[0] = ray[1] * edge2[2] - ray[2] * edge2[1];
            p[1] = ray[2] * edge2[0] - ray[0] * edge2[2];
            p[2] = ray[0] * edge2[1] - ray[1] * edge2[0];

            det = p[0] * edge1[0] + p[1] * edge1[1] + p[2] * edge1[2];
            if (det > EPSION)
            {
              vt[0] = targetRay.tar[0] - targetMesh.pnt0[0];
              vt[1] = targetRay.tar[1] - targetMesh.pnt0[1];
              vt[2] = targetRay.tar[2] - targetMesh.pnt0[2];
              u = p[0] * vt[0] + p[1] * vt[1] + p[2] * vt[2];
              if (u >= 0.0 && u <= 1.0 * det)
              {
                q[0] = vt[1] * edge1[2] - vt[2] * edge1[1];
                q[1] = vt[2] * edge1[0] - vt[0] * edge1[2];
                q[2] = vt[0] * edge1[1] - vt[1] * edge1[0];
                v = q[0] * ray[0] + q[1] * ray[1] + q[2] * ray[2];
                if (v >= 0.0 && (u + v) <= 1.0 * det)
                {
                  t = (q[0] * edge2[0] + q[1] * edge2[1] + q[2] * edge2[2]) / det;
                  u /= det;
                  v /= det;
                  pnt[0] = edge1[0] * u + edge2[0] * v + targetMesh.pnt0[0];
                  pnt[1] = edge1[1] * u + edge2[1] * v + targetMesh.pnt0[1];
                  pnt[2] = edge1[2] * u + edge2[2] * v + targetMesh.pnt0[2];
                  dist1 = sqrt((pnt[0] - targetRay.tar[0]) * (pnt[0] - targetRay.tar[0]) +
                    (pnt[1] - targetRay.tar[1]) * (pnt[1] - targetRay.tar[1]) +
                    (pnt[2] - targetRay.tar[2]) * (pnt[2] - targetRay.tar[2]));
                  if (t >= 0.0 && t <= 1.0)
                  {
                    // distance
                    float cross[3];
                    cross[0] = edge1[1] * edge2[2] - edge1[2] * edge2[1];
                    cross[1] = edge1[2] * edge2[0] - edge1[0] * edge2[2];
                    cross[2] = edge1[0] * edge2[1] - edge1[1] * edge2[0];
                    float d = -cross[0] * targetMesh.pnt0[0] - cross[1] * targetMesh.pnt0[1] - cross[2] * targetMesh.pnt0[2];
                    float dist2 = (cross[0] * pnt[0] + cross[1] * pnt[1] + cross[2] * pnt[2] + d) /
                      sqrt(cross[0] * cross[0] + cross[1] * cross[1] + cross[2] * cross[2]);
                    // output
                    _col[idx].colFlag = true;
                    _col[idx].pnt[0] = pnt[0];
                    _col[idx].pnt[1] = pnt[1];
                    _col[idx].pnt[2] = pnt[2];
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

hipError_t createLargeArrays(std::vector<Mesh> inMesh, std::vector<Ray> inRay, Mesh* outMesh, Ray* outRay)
{
  hipError_t cudaStatus;
  Mesh* dev_mesh;
  Ray* dev_ray;
  Mesh* dev_mat_mesh;
  Ray* dev_mat_ray;
  
  // input
  //Mesh* mesh = new Mesh[inMesh.size()];
  Mesh* mesh = inMesh.data();
  //for (int i = 0; i < inMesh.size(); i++)
  //{
  //  mesh[i] = inMesh[i];
  //}
  //Ray* ray = new Ray[inRay.size()];
  Ray* ray = inRay.data();
  //for (int i = 0; i < inRay.size(); i++)
  //{
  //  ray[i] = inRay[i];
  //}

  // cuda
  cudaStatus = hipMalloc((void**)&dev_mesh, inMesh.size() * sizeof(Mesh));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed_1!");
    goto Error4;
  }
  cudaStatus = hipMalloc((void**)&dev_ray, inRay.size() * sizeof(Ray));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed_2!");
    goto Error3;
  }
  cudaStatus = hipMalloc((void**)&dev_mat_mesh, inMesh.size() * inRay.size() * sizeof(Mesh));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed_1!");
    goto Error2;
  }
  cudaStatus = hipMalloc((void**)&dev_mat_ray, inMesh.size() * inRay.size() * sizeof(Ray));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed_2!");
    goto Error1;
  }

  // create stream
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Copy input vectors from host memory to GPU buffers.
  //cudaStatus = hipMemcpy(dev_mesh, mesh, cnt * sizeof(Mesh), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpyAsync(dev_mesh, mesh, inMesh.size() * sizeof(Mesh), hipMemcpyHostToDevice, stream);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed_1!");
    goto Error1;
  }
  //cudaStatus = hipMemcpy(dev_ray, ray, cnt * sizeof(Ray), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpyAsync(dev_ray, ray, inRay.size() * sizeof(Ray), hipMemcpyHostToDevice, stream);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed_2!");
    goto Error1;
  }

  // Launch a kernel on the GPU with one thread for each element.
  createLargeArray << <(inMesh.size() * inRay.size() + 1024 - 1) / 1024, 1024 >> > (inMesh.size(), inRay.size(),
    dev_mesh, dev_ray, dev_mat_mesh, dev_mat_ray);

  // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error1;
  }

  // mem copy
  cudaStatus = hipMemcpyAsync(outMesh, dev_mat_mesh, inMesh.size() * inRay.size() * sizeof(Mesh), hipMemcpyDeviceToHost, stream);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error1;
  }
  cudaStatus = hipMemcpyAsync(outRay, dev_mat_ray, inMesh.size() * inRay.size() * sizeof(Ray), hipMemcpyDeviceToHost, stream);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error1;
  }

  // stream close
  cudaStatus = hipStreamSynchronize(stream);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipStreamSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    goto Error1;
  }
  hipStreamDestroy(stream);

  //delete[] mesh;
  //delete[] ray;
Error1:
  hipFree(dev_mat_ray);
Error2:
  hipFree(dev_mat_mesh);
Error3:
  hipFree(dev_ray);
Error4:
  hipFree(dev_mesh);
  return cudaStatus;
}

__global__ void createLargeArray(int meshSize, int raySize, Mesh* inMesh, Ray* inRay, Mesh* outMesh, Ray* outRay)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = idx % meshSize;
  int iy = idx / meshSize;
  //int ix = blockIdx.x * blockDim.x + threadIdx.x;
  //int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < meshSize && iy < raySize)
  {
    //int idx = iy * meshSize + ix;
    outMesh[idx] = inMesh[ix];
    outRay[idx] = inRay[iy];
  }
}