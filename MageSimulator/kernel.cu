#include "hip/hip_runtime.h"
﻿// install nupengl.core by nuget

#include <hip/hip_runtime.h>
#include <>
#include <GL/glew.h>

#include "Structs.cuh"
#include <stdio.h>
#include "ToolPath.cuh"
#include "PrintObject.cuh"
#include "MachineObject.cuh"
#include "CollisionCheck.cuh"
#include "SimpleGL.cuh"
#include <vector>
#include <set>
#include <utility>

#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>
#include <thread>

// need install "vcpkg install tinyxml2"
#include <tinyxml2.h>
#include <omp.h>

PathItem* toolpath;
PrintItem* print;
std::vector<std::vector<PrintItem*>> largePrint;
std::vector<Ray*> largeRay;
std::vector<Mesh*> largeMesh;
int toolpathFullCnt;
int printFullCnt;
std::vector<int> printHeader;
bool isloop = true;
int timeCnt = 0;
Kinematics* kin;
std::vector<CollisionObject*> arrBodies;
std::vector<CollisionObject*> arrBeds;
bool enableCheckCollsion = false;
bool enableMultiThread = false;
std::vector<CollisionSet>* resultCol;

GLuint pbo;
hipGraphicsResource* dev_resource;

std::vector<CollisionSet> checkTime(int time, bool isdraw)
{
	clock_t time1 = clock();
	std::vector<CollisionSet> retcol = std::vector<CollisionSet>();
	// large
	std::vector<Ray> largeRay = std::vector<Ray>();
	std::vector<std::vector<Mesh>> bodyMeshBuffer = std::vector<std::vector<Mesh>>();
	bool** targetGroup = new bool* [arrBodies.size()];
	for (int m = 0; m < arrBodies.size(); m++)
	{
		targetGroup[m] = new bool[largeMesh.size()];
		for (int i = 0; i < largeMesh.size(); i++)
		{
			targetGroup[m][i] = false;
		}
		std::vector<Mesh> arrBodyMesh = arrBodies[m]->move(print[time], kin, isdraw);
		bodyMeshBuffer.push_back(arrBodyMesh);
		for (int n = 0; n < arrBodyMesh.size(); n++)
		{
			Ray buf0;
			buf0.org[0] = arrBodyMesh[n].pnt0[0];
			buf0.org[1] = arrBodyMesh[n].pnt0[1];
			buf0.org[2] = arrBodyMesh[n].pnt0[2];
			buf0.tar[0] = arrBodyMesh[n].pnt1[0];
			buf0.tar[1] = arrBodyMesh[n].pnt1[1];
			buf0.tar[2] = arrBodyMesh[n].pnt1[2];
			buf0.id = m;
			largeRay.push_back(buf0);
			Ray buf1;
			buf1.org[0] = arrBodyMesh[n].pnt1[0];
			buf1.org[1] = arrBodyMesh[n].pnt1[1];
			buf1.org[2] = arrBodyMesh[n].pnt1[2];
			buf1.tar[0] = arrBodyMesh[n].pnt2[0];
			buf1.tar[1] = arrBodyMesh[n].pnt2[1];
			buf1.tar[2] = arrBodyMesh[n].pnt2[2];
			buf1.id = m;
			largeRay.push_back(buf1);
			Ray buf2;
			buf2.org[0] = arrBodyMesh[n].pnt2[0];
			buf2.org[1] = arrBodyMesh[n].pnt2[1];
			buf2.org[2] = arrBodyMesh[n].pnt2[2];
			buf2.tar[0] = arrBodyMesh[n].pnt0[0];
			buf2.tar[1] = arrBodyMesh[n].pnt0[1];
			buf2.tar[2] = arrBodyMesh[n].pnt0[2];
			buf2.id = m;
			largeRay.push_back(buf2);
		}
	}
	clock_t time7 = clock();
	std::vector<Mesh> largeTargetMesh = std::vector<Mesh>();
	for (int j = 0; j < largeMesh.size(); j++)
	{
		for (int k = 0; k < 12; k++)
		{
			largeTargetMesh.push_back(largeMesh[j][k]);
		}
	}
	//std::vector<Mesh>* bedMeshBuffer = new std::vector<Mesh>[largeRay.size()];
	std::vector<Mesh>* bedMeshBuffer = new std::vector<Mesh>[arrBeds.size()];
	for (int j = 0; j < arrBeds.size(); j++)
	{
		std::vector<Mesh> arrBedMesh = arrBeds[j]->move(print[time], kin, isdraw);
		bedMeshBuffer[j] = arrBedMesh;
		Mesh* bboxMesh = createMesh2Bbox(arrBedMesh, largeMesh.size() + j);
		for (int k = 0; k < 12; k++)
		{
			largeTargetMesh.push_back(bboxMesh[k]);
		}
	}
	Mesh* bufLargeMesh;
	bufLargeMesh = (Mesh*)malloc(sizeof(Mesh) * largeTargetMesh.size() * largeRay.size());
	Ray* bufLargeRay;
	bufLargeRay = (Ray*)malloc(sizeof(Ray) * largeTargetMesh.size() * largeRay.size());
	clock_t time9 = clock();
	createLargeArrays(largeTargetMesh, largeRay, bufLargeMesh, bufLargeRay);

	clock_t time2 = clock();
	CollisionSet* largeCol = (CollisionSet*)malloc(sizeof(CollisionSet) * largeTargetMesh.size() * largeRay.size());
	clock_t time3 = clock();
	checkCollisions(largeTargetMesh.size() * largeRay.size(), bufLargeMesh, bufLargeRay, largeCol);
	clock_t time8 = clock();
	std::vector<std::pair<int, int>> pairBodyGroup = std::vector<std::pair<int, int>>();
	for (int i = 0; i < largeTargetMesh.size() * largeRay.size(); i++)
	{
		if (largeCol[i].colFlag)
		{
			std::pair<int, int > pair = std::pair<int, int>();
			pair.first = largeCol[i].rayId;
			pair.second = largeCol[i].meshId;
			pairBodyGroup.push_back(pair);
		}
	}
	std::sort(pairBodyGroup.begin(), pairBodyGroup.end());
	auto last = std::unique(pairBodyGroup.begin(), pairBodyGroup.end());
	pairBodyGroup.erase(last, pairBodyGroup.end());
	// small
	std::vector<Mesh> smallMesh = std::vector<Mesh>();
	std::vector<Ray> smallRay = std::vector<Ray>();
	for (int i = 0; i < pairBodyGroup.size(); i++)
	{
		int body = pairBodyGroup[i].first;
		int group = pairBodyGroup[i].second;
		for (int j = 0; j < bodyMeshBuffer[body].size(); j++)
		{
			if (group < largeMesh.size())
			{
				for (int k = 0; k < largePrint[group].size(); k++)
				{
					if (largePrint[group][k]->time < time)
					{
						for (int m = 0; m < 12; m++)
						{
							smallMesh.push_back(bodyMeshBuffer[body][j]);
							smallRay.push_back(largePrint[group][k]->ray[m]);
						}
					}
				}
			}
			else
			{
				int idx = group - largeMesh.size();
				for (int m = 0; m < bedMeshBuffer[idx].size(); m++)
				{
					Ray buf0;
					buf0.org[0] = bedMeshBuffer[idx][m].pnt0[0];
					buf0.org[1] = bedMeshBuffer[idx][m].pnt0[1];
					buf0.org[2] = bedMeshBuffer[idx][m].pnt0[2];
					buf0.tar[0] = bedMeshBuffer[idx][m].pnt1[0];
					buf0.tar[1] = bedMeshBuffer[idx][m].pnt1[1];
					buf0.tar[2] = bedMeshBuffer[idx][m].pnt1[2];
					buf0.id = group;
					smallMesh.push_back(bodyMeshBuffer[body][j]);
					smallRay.push_back(buf0);
					Ray buf1;
					buf1.org[0] = bedMeshBuffer[idx][m].pnt1[0];
					buf1.org[1] = bedMeshBuffer[idx][m].pnt1[1];
					buf1.org[2] = bedMeshBuffer[idx][m].pnt1[2];
					buf1.tar[0] = bedMeshBuffer[idx][m].pnt2[0];
					buf1.tar[1] = bedMeshBuffer[idx][m].pnt2[1];
					buf1.tar[2] = bedMeshBuffer[idx][m].pnt2[2];
					buf1.id = group;
					smallMesh.push_back(bodyMeshBuffer[body][j]);
					smallRay.push_back(buf1);
					Ray buf2;
					buf2.org[0] = bedMeshBuffer[idx][m].pnt2[0];
					buf2.org[1] = bedMeshBuffer[idx][m].pnt2[1];
					buf2.org[2] = bedMeshBuffer[idx][m].pnt2[2];
					buf2.tar[0] = bedMeshBuffer[idx][m].pnt0[0];
					buf2.tar[1] = bedMeshBuffer[idx][m].pnt0[1];
					buf2.tar[2] = bedMeshBuffer[idx][m].pnt0[2];
					buf2.id = group;
					smallMesh.push_back(bodyMeshBuffer[body][j]);
					smallRay.push_back(buf2);
				}
			}
		}
	}
	Mesh* bufSmallMesh;
	bufSmallMesh = (Mesh*)malloc(sizeof(Mesh) * smallMesh.size());
	Ray* bufSmallRay;
	bufSmallRay = (Ray*)malloc(sizeof(Ray) * smallMesh.size());
	for (int i = 0; i < smallMesh.size(); i++)
	{
		bufSmallMesh[i] = smallMesh[i];
		bufSmallRay[i] = smallRay[i];
	}
	clock_t time4 = clock();
	CollisionSet* smallCol = (CollisionSet*)malloc(sizeof(CollisionSet) * smallMesh.size());
	clock_t time5 = clock();
	checkCollisions(smallMesh.size(), bufSmallMesh, bufSmallRay, smallCol);
	for (int i = 0; i < smallMesh.size(); i++)
	{
		if (smallCol[i].colFlag)
		{
			retcol.push_back(smallCol[i]);
		}
	}

	// memory clear
	free(bufLargeMesh);
	free(bufLargeRay);
	free(largeCol);
	largeRay.clear();
	pairBodyGroup.clear();
	smallMesh.clear();
	smallRay.clear();
	bodyMeshBuffer.clear();
	free(bufSmallMesh);
	free(bufSmallRay);
	free(smallCol);
	clock_t time6 = clock();
	float time17 = static_cast<float>(time7 - time1) / CLOCKS_PER_SEC * 1000.0;
	float time79 = static_cast<float>(time9 - time7) / CLOCKS_PER_SEC * 1000.0;
	float time92 = static_cast<float>(time2 - time9) / CLOCKS_PER_SEC * 1000.0;
	float time23 = static_cast<float>(time3 - time2) / CLOCKS_PER_SEC * 1000.0;
	float time38 = static_cast<float>(time8 - time3) / CLOCKS_PER_SEC * 1000.0;
	float time84 = static_cast<float>(time4 - time8) / CLOCKS_PER_SEC * 1000.0;
	float time45 = static_cast<float>(time5 - time4) / CLOCKS_PER_SEC * 1000.0;
	float time56 = static_cast<float>(time6 - time5) / CLOCKS_PER_SEC * 1000.0;
	//printf("%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", time17, time79, time92, time23, time38, time84, time45, time56);

	return retcol;
}

void sampling()
{
	if (isloop)
	{
		// drawBox
		float time_gl = 0.0;
		if (timeCnt % 10 == 0)
		{
			extern CameraParam cameraParam;
			kin->DisplayStart(cameraParam, print[timeCnt]);
			drawCoordinate(50.0, print[timeCnt], kin);
			clock_t start_gl = clock();
			for (int i = 0; i < timeCnt; i++)
			{
				if (print[i].type == PrintType::OuterWall || print[i].type == PrintType::Infill || print[i].type == PrintType::Print)
				{
					float h = i / (float)printFullCnt * 360.0;
					float faceR, faceG, faceB, edgeR, edgeG, edgeB;
					HSVtoRGB(h, 1.0, 1.0, faceR, faceG, faceB);
					HSVtoRGB(h, 0.5, 1.0, edgeR, edgeG, edgeB);
					drawPrintItem_box(print[i], faceR, faceG, faceB, edgeR, edgeG, edgeB);
				}
				else
				{
					drawPrintItem_move(print[i], 1.0, 1.0, 1.0);
				}
			}
			clock_t end_gl = clock();
			time_gl = static_cast<float>(end_gl - start_gl) / CLOCKS_PER_SEC * 1000.0;
		}

		// time 
		clock_t start_cuda = clock();
		// collision check
		std::vector<CollisionSet> colset;
		if (enableCheckCollsion)
		{
			colset = checkTime(timeCnt, true);
			for (int i = 0; i < colset.size(); i++)
			{
				glColor3d(0.5, 0.5, 0.5);
				glPushMatrix();
				glTranslatef(colset[i].pnt[0], colset[i].pnt[1], colset[i].pnt[2]);
				glutSolidSphere(1.0, 10, 10);
				glPopMatrix();
			}
		}
		else
		{
			for (int m = 0; m < arrBodies.size(); m++)
			{
				std::vector<Mesh> arrBodyMesh = arrBodies[m]->move(print[timeCnt], kin, true);
			}
			for (int j = 0; j < arrBeds.size(); j++)
			{
				std::vector<Mesh> arrBedMesh = arrBeds[j]->move(print[timeCnt], kin, true);
			}
		}

		// time
		clock_t end_cuda = clock();
		float time_cuda = static_cast<float>(end_cuda - start_cuda) / CLOCKS_PER_SEC * 1000.0;
		if (enableCheckCollsion)
		{
			printf("time:%d,line:%d,block:%d,layer:%d,collision:%d\n",
				timeCnt, print[timeCnt].lineIdx, print[timeCnt].blockIdx, print[timeCnt].layerIdx, colset.size());
		}
		else
		{
			printf("time:%d,line:%d,block:%d,layer:%d\n",
				timeCnt, print[timeCnt].lineIdx, print[timeCnt].blockIdx, print[timeCnt].layerIdx);
		}

		// draw and next time
		if (timeCnt % 10 == 0)
		{
			glutSwapBuffers();
		}
		timeCnt++;
		if (timeCnt >= printFullCnt) timeCnt = printFullCnt - 1;
	}
	else
	{
		extern CameraParam cameraParam;
		kin->DisplayStart(cameraParam, print[timeCnt]);
		drawCoordinate(50.0, print[timeCnt], kin);
		for (int i = 0; i < timeCnt; i++)
		{
			if (print[i].type == PrintType::OuterWall || print[i].type == PrintType::Infill || print[i].type == PrintType::Print)
			{
				float h = i / (float)printFullCnt * 360.0;
				float faceR, faceG, faceB, edgeR, edgeG, edgeB;
				HSVtoRGB(h, 1.0, 1.0, faceR, faceG, faceB);
				HSVtoRGB(h, 0.5, 1.0, edgeR, edgeG, edgeB);
				drawPrintItem_box(print[i], faceR, faceG, faceB, edgeR, edgeG, edgeB);
			}
			else
			{
				drawPrintItem_move(print[i], 1.0, 1.0, 1.0);
			}
		}

		for (int m = 0; m < arrBodies.size(); m++)
		{
			std::vector<Mesh> arrBodyMesh = arrBodies[m]->move(print[timeCnt], kin, true);
		}
		for (int j = 0; j < arrBeds.size(); j++)
		{
			std::vector<Mesh> arrBedMesh = arrBeds[j]->move(print[timeCnt], kin, true);
		}
		glutSwapBuffers();
	}
}

int main(int argc, char** argv)
{
	// read param
	// 1:gcode
	//char* filepath = "C:\\Develop\\Slicer\\slicer_models\\output_viewer.gcode";
	char* filepath = argv[1];
	printf("GcodeFile:%s\n", filepath);
	// 2:machine
	char* machinefile = argv[2];
	printf("MachineFile:%s\n", machinefile);
	char* strCol = argv[3];
	enableCheckCollsion = false;
	if (strcmp(strCol, "true") == 0)
	{
		enableCheckCollsion = true;
	}
	printf("enableCheckCollision:%s\n", (enableCheckCollsion ? "true" : "false"));
	char* strMulti = argv[4];
	enableMultiThread = false;
	if (strcmp(strMulti, "true") == 0)
	{
		enableMultiThread = true;
	}
	printf("enableMultiThread:%s\n", (enableMultiThread ? "true" : "false"));

	// read machine xml
	tinyxml2::XMLDocument doc;
	tinyxml2::XMLError err = doc.LoadFile(machinefile);
	tinyxml2::XMLElement* xmlRoot = doc.FirstChildElement("Machine");
	tinyxml2::XMLElement* xmlKinematics = xmlRoot->FirstChildElement("Kinematics");
	printf("Kinematics:%s", xmlKinematics->GetText());
	if (strcmp(xmlKinematics->GetText(), "CoreXY-BC") == 0)
	{
		kin = new CoreXYBC();
	}
	else if (strcmp(xmlKinematics->GetText(), "CoreXY") == 0)
	{
		kin = new CoreXY();
	}
	else if (strcmp(xmlKinematics->GetText(), "BedSlingerY") == 0)
	{
		kin = new BedSlingerY();
	}
	else if (strcmp(xmlKinematics->GetText(), "Delta") == 0)
	{
		kin = new Delta();
	}
	else
	{
		printf("Unknown Kinematics has been selected.");
		exit(-1);
	}
	arrBodies = std::vector<CollisionObject*>();
	arrBeds = std::vector<CollisionObject*>();
	// head
	tinyxml2::XMLElement* xmlHeads = xmlRoot->FirstChildElement("Heads");
	if (xmlHeads)
	{
		tinyxml2::XMLElement* xmlHead = xmlHeads->FirstChildElement("Head");
		while (xmlHead)
		{
			tinyxml2::XMLElement* xmlType = xmlHead->FirstChildElement("Type");
			if (strcmp(xmlType->GetText(), "Cylinder") == 0)
			{
				CollisionCylinder* cylinder = new CollisionCylinder(MachineType::Head, xmlHead);
				arrBodies.push_back(cylinder);
			}
			else if (strcmp(xmlType->GetText(), "Box") == 0)
			{
				CollisionBox* box = new CollisionBox(MachineType::Head, xmlHead);
				arrBodies.push_back(box);
			}
			xmlHead = xmlHead->NextSiblingElement("Head");
		}
	}
	// xgantry
	tinyxml2::XMLElement* xmlXGantrys = xmlRoot->FirstChildElement("XGantrys");
	if (xmlXGantrys)
	{
		tinyxml2::XMLElement* xmlXGantry = xmlXGantrys->FirstChildElement("XGantry");
		while (xmlXGantry)
		{
			tinyxml2::XMLElement* xmlType = xmlXGantry->FirstChildElement("Type");
			if (strcmp(xmlType->GetText(), "Cylinder") == 0)
			{
				CollisionCylinder* cylinder = new CollisionCylinder(MachineType::XGantry, xmlXGantry);
				arrBodies.push_back(cylinder);
			}
			else if (strcmp(xmlType->GetText(), "Box") == 0)
			{
				CollisionBox* box = new CollisionBox(MachineType::XGantry, xmlXGantry);
				arrBodies.push_back(box);
			}
			xmlXGantry = xmlXGantry->NextSiblingElement("XGantry");
		}
	}
	// ygantry
	tinyxml2::XMLElement* xmlYGantrys = xmlRoot->FirstChildElement("YGantrys");
	if (xmlYGantrys)
	{
		tinyxml2::XMLElement* xmlYGantry = xmlYGantrys->FirstChildElement("YGantry");
		while (xmlYGantry)
		{
			tinyxml2::XMLElement* xmlType = xmlYGantry->FirstChildElement("Type");
			if (strcmp(xmlType->GetText(), "Cylinder") == 0)
			{
				CollisionCylinder* cylinder = new CollisionCylinder(MachineType::YGantry, xmlYGantry);
				arrBodies.push_back(cylinder);
			}
			else if (strcmp(xmlType->GetText(), "Box") == 0)
			{
				CollisionBox* box = new CollisionBox(MachineType::YGantry, xmlYGantry);
				arrBodies.push_back(box);
			}
			xmlYGantry = xmlYGantry->NextSiblingElement("YGantry");
		}
	}
	// bed
	tinyxml2::XMLElement* xmlBeds = xmlRoot->FirstChildElement("Beds");
	if (xmlBeds)
	{
		tinyxml2::XMLElement* xmlBed = xmlBeds->FirstChildElement("Bed");
		while (xmlBed)
		{
			tinyxml2::XMLElement* xmlType = xmlBed->FirstChildElement("Type");
			if (strcmp(xmlType->GetText(), "Cylinder") == 0)
			{
				CollisionCylinder* cylinder = new CollisionCylinder(MachineType::Bed, xmlBed);
				arrBeds.push_back(cylinder);
			}
			else if (strcmp(xmlType->GetText(), "Box") == 0)
			{
				CollisionBox* box = new CollisionBox(MachineType::Bed, xmlBed);
				arrBeds.push_back(box);
			}
			xmlBed = xmlBed->NextSiblingElement("Bed");
		}
	}

	// read gcode
	std::vector<GcodeItem> bufgcode = readGcode(filepath);
	int gcodeCnt = bufgcode.size();
	std::vector<PathItem> bufpath = convertGcode2Path(kin, bufgcode);

  // read csv
	toolpathFullCnt = bufpath.size();
  toolpath = (PathItem*)malloc(sizeof(PathItem) * toolpathFullCnt);
  for (int i = 0; i < toolpathFullCnt; i++)
  {
    toolpath[i] = bufpath[i];
  }
	printFullCnt = toolpathFullCnt - 1;
  print = (PrintItem*)malloc(sizeof(PrintItem) * printFullCnt);
  hipError_t cudaStatus = calcPrintItems(toolpathFullCnt, toolpath, print);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "addWithCuda failed!");
    return 1;
  }
	printf("read file end, step count:%d\n", printFullCnt);

	// header check
	int preBlock = -1;
	int preLayer = -1;
	printHeader = std::vector<int>();
	for (int i = 0; i < printFullCnt; i++)
	{
		int block = print[i].blockIdx;
		int layer = print[i].layerIdx;
		if (preBlock != block || preLayer != layer)
		{
			printHeader.push_back(i);
		}
		preBlock = block;
		preLayer = layer;
	}

	std::vector<PrintItem*> bufPrint;
	for (int i = 0; i < printFullCnt; i++)
	{
		bufPrint.push_back(&print[i]);
	}
	largePrint = std::vector<std::vector<PrintItem*>>();
	largeRay = std::vector<Ray*>();
	largeMesh = std::vector<Mesh*>();
	constructBVH(bufPrint, &largePrint, &largeRay, &largeMesh);

	if (enableMultiThread)
	{
		// time split
		int numThread = std::thread::hardware_concurrency();
		//int numThread = 1;
		printf("NumberThread:%d\n", numThread);
		printf("Start multi thread\n");
		
		std::vector<int> divTimeFirst = std::vector<int>();
		std::vector<int> divTimeLast = std::vector<int>();
		divTimeFirst.push_back(0);
		for (int i = 0; i < numThread; i++)
		{
			float val = (float)printFullCnt * (float)printFullCnt / (float)numThread + (float)divTimeFirst[i] * (float)divTimeFirst[i];
			int bufval = sqrt(val);
			if (i == numThread - 1)
			{
				bufval = printFullCnt - 1;
			}
			divTimeLast.push_back(bufval);
			printf("%d,%d,%d\n", i, divTimeFirst[i], divTimeLast[i] - divTimeFirst[i]);
			divTimeFirst.push_back(bufval);
		}
		resultCol = new std::vector<CollisionSet>[printFullCnt];

#pragma omp parallel for schedule(static)
		for (int i = 0; i < numThread; i++)
		{
			for (int j = divTimeFirst[i]; j < divTimeLast[i]; j++)
			{
				clock_t start_cuda = clock();
				std::vector<CollisionSet> colset = checkTime(j, false);
				resultCol[j] = colset;
				clock_t end_cuda = clock();
				float time_cuda = static_cast<float>(end_cuda - start_cuda) / CLOCKS_PER_SEC * 1000.0;
#pragma omp critical
				{
					printf("%d,%d,%d,%lf\n", i, j, colset.size(), time_cuda);
				}
			}
		}
#pragma omp barrier
		
		printf("End multi thread\n");
	}
	else
	{
		// opengl
		init(argc, argv);
	}

  return 0;
}
